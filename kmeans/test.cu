/*
Copyright 2013  Bryan Catanzaro

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
kmeans
*/
#include <thrust/device_vector.h>
#include "kmeans.h"
#include "timer.h"
#include <iostream>
#include <cstdlib>
#include <typeinfo>
#include "test.h"


template<typename T>
void print_array(T& array, int m, int n) {
    for(int i = 0; i < m; i++) {
        for(int j = 0; j < n; j++) {
            typename T::value_type value = array[i * n + j];
            std::cout << value << " ";
        }
        std::cout << std::endl;
    }
}

template<typename T>
void fill_array(T& array, int m, int n) {
    for(int i = 0; i < m; i++) {
        for(int j = 0; j < n; j++) {
            array[i * n + j] = (i % 2)*3 + j;
        }
    }
}

template<typename T>
void random_data(thrust::device_vector<T>& array, int m, int n) {
    thrust::host_vector<T> host_array(m*n);
    for(int i = 0; i < m * n; i++) {
        host_array[i] = (T)rand()/(T)RAND_MAX;
    }
    array = host_array;
}

void random_labels(thrust::device_vector<int>& labels, int n, int k) {
    thrust::host_vector<int> host_labels(n);
    for(int i = 0; i < n; i++) {
        host_labels[i] = rand() % k;
    }
    labels = host_labels;
}


void tiny_test() {
    int iterations = 1;
    int n = 5;
    int d = 3;
    int k = 2;

    
    thrust::device_vector<double> data(n * d);
    thrust::device_vector<int> labels(n);
    thrust::device_vector<double> centroids(k * d);
    thrust::device_vector<double> distances(n);
    
    fill_array(data, n, d);
    std::cout << "Data: " << std::endl;
    print_array(data, n, d);

    labels[0] = 0;
    labels[1] = 0;
    labels[2] = 0;
    labels[3] = 1;
    labels[4] = 1;

    std::cout << "Labels: " << std::endl;
    print_array(labels, n, 1);
    
    int i = kmeans::kmeans(iterations, n, d, k, data, labels, centroids, distances);
    std::cout << "Performed " << i << " iterations" << std::endl;

    std::cout << "Labels: " << std::endl;
    print_array(labels, n, 1);

    std::cout << "Centroids:" << std::endl;
    print_array(centroids, k, d);

    std::cout << "Distances:" << std::endl;
    print_array(distances, n, 1);

}


void more_tiny_test() {
	double dataset[] = {
		0.5, 0.5,
		1.5, 0.5,
		1.5, 1.5,
		0.5, 1.5,
		1.1, 1.2,
		0.5, 15.5,
		1.5, 15.5,
		1.5, 16.5,
		0.5, 16.5,
		1.2, 16.1,
		15.5, 15.5,
		16.5, 15.5,
		16.5, 16.5,
		15.5, 16.5,
		15.6, 16.2,
		15.5, 0.5,
		16.5, 0.5,
		16.5, 1.5,
		15.5, 1.5,
		15.7, 1.6};
	double centers[] = {
		0.5, 0.5,
		1.5, 0.5,
		1.5, 1.5,
		0.5, 1.5};
	 
    int iterations = 3;
    int n = 20;
    int d = 2;
    int k = 4;
	
    thrust::device_vector<double> data(dataset, dataset+n*d);
    thrust::device_vector<int> labels(n);
    thrust::device_vector<double> centroids(centers, centers+k*d);
    thrust::device_vector<double> distances(n);
    
    kmeans::kmeans(iterations, n, d, k, data, labels, centroids, distances, false);

    std::cout << "Labels: " << std::endl;
    print_array(labels, n, 1);

    std::cout << "Centroids:" << std::endl;
    print_array(centroids, k, d);

}

template<typename T> void huge_test(void) {

    int iterations = 50;
    int n = 1e6;
    int d = 64;
    int k = 128;

    thrust::device_vector<T> data(n * d);
    thrust::device_vector<int> labels(n);
    thrust::device_vector<T> centroids(k * d);
    thrust::device_vector<T> distances(n);
    
    std::cout << "Generating random data" << std::endl;
    std::cout << "Number of points: " << n << std::endl;
    std::cout << "Number of dimensions: " << d << std::endl;
    std::cout << "Number of clusters: " << k << std::endl;
    std::cout << "Number of iterations: " << iterations << std::endl;
    std::cout << "Precision: " << typeid(T).name() << std::endl;
    
    random_data(data, n, d);
    random_labels(labels, n, k);
    kmeans::timer t;
    t.start();
    kmeans::kmeans(iterations, n, d, k, data, labels, centroids, distances);
    float time = t.stop();
    std::cout << "  Time: " << time/1000.0 << " s" << std::endl;

}


//This is a wrapper function written by Muralikrishna 
// to use the tempalte function which invokes the 
// double type of template function 
void huge_test_double()
{
  huge_test<double>();
}


// Wrapper function to call the CUDA KMeans algorithm
// Written by Muralikrishna SN
void cudaKmeans(int iterations,int n, int d,int  k, float *dataset,int *labels,float *centers)
{

      // Call parallel kmeans

        thrust::device_vector<float> data(dataset, dataset+n*d);
        thrust::device_vector<int> Labels(n);
        thrust::device_vector<float> centroids(centers, centers+k*d);
        thrust::device_vector<float> distances(n);

        kmeans::kmeans(iterations, n, d, k, data, Labels, centroids, distances,false);

//        thrust::host_vector <int> resultLabel(Labels.begin(), Labels.end() ); 
//        thrust::host_vector <float> newCentroids(centroids.begin(), centroids.end() );
        for(int i = 0; i < Labels.size(); i++)
                 labels[i]=Labels[i];
        for(int i = 0; i < centroids.size(); i++)
                 centers[i]=centroids[i];
         
}


/*
int main() {
    std::cout << "Input a character to choose a test:" << std::endl;
    std::cout << "Tiny test: t" << std::endl;
    std::cout << "More tiny test: m" << std::endl;
    std::cout << "Huge test: h: " << std::endl;
    char c;
    std::cin >> c;
    switch (c) {
    case 't':
        tiny_test();
        exit(0);
    case 'm':
        more_tiny_test();
        exit(0);
    case 'h':
        break;
    default:
        std::cout << "Choice not understood, running huge test" << std::endl;
    }
    std::cout << "Double precision (d) or single precision (f): " << std::endl;
    std::cin >> c;
    switch(c) {
    case 'd':
        huge_test<double>();
        exit(0);
    case 'f':
        break;
    default:
        std::cout << "Choice not understood, running single precision"
                  << std::endl;
    }
    huge_test<float>();
    
}*/
